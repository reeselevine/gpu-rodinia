#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL2_H_
#define _KERNEL2_H_

#include <cuda/atomic>

__global__ void
Kernel2( cuda::atomic<bool, cuda::thread_scope_device>* g_graph_mask, cuda::atomic<bool, cuda::thread_scope_device> *g_updating_graph_mask, bool* g_graph_visited, bool *g_over, int no_of_nodes)
{
	cuda::memory_order mem_order = cuda::memory_order_relaxed; 
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if( tid<no_of_nodes && g_updating_graph_mask[tid].load(mem_order))
	{

		g_graph_mask[tid].store(true, mem_order);
		g_graph_visited[tid]=true;
		*g_over=true;
		g_updating_graph_mask[tid].store(false, mem_order);
	}
}

#endif

// makeAtomic:
// g_updating_graph_mask: control dependency (25), written to (31)

